#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <iomanip>
#include <cstring>

using namespace std;
typedef vector<vector<int>> matrix;

struct calculo
{
  thrust::device_ptr<char> d_S;
  char letradeT;
  thrust::device_ptr<int> calc;

  calculo(thrust::device_ptr<char> d_S_, char letradeT_, thrust::device_ptr<int> calc_)
      : d_S(d_S_),
        letradeT(letradeT_),
        calc(calc_)
  {
  }
  // essas marcações indicam que o código deve ser compilado para CPU (host)
  // e GPU (device)
  // IMPORTANTE: somente código com a marcação __device__ é compilado para GPU
  __host__ __device__

      int
      operator()(const int &i)
  {
    int w;
    int diagonal;
    int insertion;
    int max = 0;
    if (d_S[i - 1] == letradeT)
    {
      w = 2;
    }
    else
    {
      w = -1;
    }

    diagonal = calc[i - 1] + w;
    insertion = calc[i] - 1;

    if (diagonal > max)
    {
      max = diagonal;
    }
    if (insertion > max)
    {
      max = insertion;
    }

    return max;
  }
};
struct inline_score
{

  __host__ __device__ int
  operator()(const int &i, const int &j)
  {
    int actual = j;
    int left = i - 1;
    int max;

    if (left >= actual)
      max = left;
    else
      max = actual;

    if (max < 0)
      max = 0;

    return max;
  }
};
vector<string> subGenerator(int size, string seq)
{
  vector<string> sub_seqs;
  for (int i = 0; i < size; i++)
  {
    string cand = "";
    for (int j = i; j < size; j++)
    {
      cand += seq[j];
      sub_seqs.push_back(cand);
    }
  }

  return sub_seqs;
}

matrix high_score(string a, string b, int n, int m, matrix H)
{
  int diagonal, deletion, insertion, w;

  for (int i = 1; i <= n; i++)
  {
    for (int j = 1; j <= m; j++)
    {

      if (a[i] == b[j])
      {
        w = 2;
      }
      else
      {
        w = -1;
      }

      diagonal = H[i - 1][j - 1] + w;
      deletion = H[i - 1][j] - 1;
      insertion = H[i][j - 1] - 1;

      H[i][j] = max({0, diagonal, deletion, insertion});
    }
  }

  return H;
}

int main()
{

  int N, M, score, max = 0;
  string S, T, seq1, seq2, seq1Def, seq2Def;
  vector<string> subs1, subs2;
  matrix H;

  cin >> N >> M;
  cin >> S >> T;

  // cout << "seq1 size: " << n << endl;
  // cout << "seq2 size: " << m << endl;
  // cout << "seq1 is: " << seq1 << endl;
  // cout << "seq2 is: " << seq2 << endl;

  // subs1 = subGenerator(n, seq1);

  // thrust::device_vector<string> sub_device(subs1);
  // thrust::copy(subs1.begin(), subs1.end(), sub_vector1.begin());

  // subs2 = subGenerator(m, seq2);
  // thrust::copy(thrust::device, subs2.begin(), subs2.end(), sub_vector2.begin());

  // for (int i = 0; i < int(subs1.size()); i++)
  // {
  //   for (int j = 0; j < int(subs2.size()); j++)
  //   {

  //     string S = subs1[i];
  //     string T = subs2[j];

  // string S = "ACT";
  // string T = "GCT";

  // const char *L = S.c_str();

  // int L = strlen(S);
  // int M = strlen(T);

  // device_vector<int> calc[2];
  // calc[0].resize(N + 1);
  // calc[1].resize(N + 1);

  // sub1.insert(0, 1, '-');
  // sub2.insert(0, 1, '-');

  // int N = S.size();
  // int M = T.size();

  thrust::device_vector<int> calc(N + 1);
  thrust::device_vector<int> temp(N + 1);

  thrust::fill(calc.begin(), calc.end(), 0);
  thrust::fill(temp.begin(), temp.end(), 0);

  thrust::device_vector<char> d_S(S.begin(), S.end());

  thrust::counting_iterator<int> c0(1);
  thrust::counting_iterator<int> c1(N + 1);

  for (int j = 0; j < M; j++)
  {
    char letradeT = T[j];
    // cout << d_S[1] << endl;
    thrust::transform(c0, c1, temp.begin() + 1, calculo(d_S.data(), letradeT, calc.data()));
    thrust::inclusive_scan(temp.begin() + 1, temp.end(), calc.begin() + 1, inline_score());
    // cout << "calc: " << endl;
    for (int l = 0; l < calc.size(); l++)
    {
      cout << " " << calc[l];
    }
    score = thrust::reduce(calc.begin(), calc.end(), 0, thrust::maximum<int>());

    if (score > max)
    {
      max = score;
    }
    // cout << endl;
    // cout << "temp: " << endl;
    // for (int l = 0; l < temp.size(); l++)
    // {
    //   cout << " " << temp[l];
    // }
    cout << endl;
  }

  // thrust::copy(d_S.begin(), d_S.end(), std::ostream_iterator<char>(std::cout, " "));

  // calc.begin().resize(N + 1);
  // H.resize(size_sub1 + 1);
  // for (int e = 0; e <= size_sub1; e++)
  // {
  //   H[e].resize(size_sub2 + 1);
  // }

  // H = high_score(sub1, sub2, size_sub1, size_sub2, H);

  // for (int i = 1; i <= size_sub1; i++)
  // {
  //   for (int j = 1; j <= size_sub2; j++)
  //   {
  //     if (H[i][j] > max)
  //     {
  //       max = H[i][j];
  //       seq1Def = sub1;
  //       seq2Def = sub2;
  //     }
  //   }
  // }
  // }
  cout << "Score Maximo: " << max << endl;
  return 0;
}
// cout << endl;
// cout << "Sub Seq1 Best: " << seq1Def.substr(1, sizeof(seq1Def)) << endl;
// cout << "Sub Seq2 Best: " << seq2Def.substr(1, sizeof(seq2Def)) << endl;
// }