#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <iomanip>
#include <cstring>

using namespace std;

struct calculo
{
    thrust::device_ptr<char> vA;
    thrust::device_ptr<char> vB;

    calculo(thrust::device_ptr<char> vA_, thrust::device_ptr<char> vB_)
        : vA(vA_),
          vB(vB_)
    {
    }
    __host__ __device__

        int
        operator()(const int &i, const int &j)
    {
        int w = -1;
        if (vA[i] == vB[j])
        {
            w = 2;
        }

        return w;
    }
};

// struct inline_score
// {

//     __host__ __device__ int
//     operator()(const int &i, const int &j)
//     {
//         int actual = j;
//         int left = i - 1;
//         int max;

//         if (left >= actual)
//             max = left;
//         else
//             max = actual;

//         if (max < 0)
//             max = 0;

//         return max;
//     }
// };

int main()
{

    int n, m, max, endA, endB, sizeSubA, sizeSubB, tam;
    string seq1, seq2;

    cin >> n >> m;
    cin >> seq1 >> seq2;

    thrust::device_vector<char> vA(seq1.begin(), seq1.end());
    thrust::device_vector<char> vB(seq2.begin(), seq2.end());

    if (seq1.size() > seq2.size())
    {
        tam = seq1.size();
    }
    else
    {
        tam = seq2.size();
    }

    thrust::device_vector<int> ponto(tam);

    cout << "seq1 size: " << n << endl;
    cout << "seq2 size: " << m << endl;
    cout << "seq1 is: " << seq1 << endl;
    cout << "seq2 is: " << seq2 << endl;

    for (int beginA = 0; beginA < vA.size(); beginA++)
    {
        for (int beginB = 0; beginB < vB.size(); beginB++)
        {
            endA = beginA;
            while (endA < vA.size())
            {
                endB = beginB;
                sizeSubA = endA - beginA;
                endA++;
                while (endB < vB.size())
                {
                    sizeSubB = endB - beginB;
                    if (sizeSubA == sizeSubB && beginB + sizeSubA < vB.size())
                    {
                        int l = 0;
                        // cout << "beginA: " << beginA << endl;
                        // cout << "endA: " << endA << endl
                        //      << endl;
                        // cout << "beginB: " << beginB << endl;
                        // cout << "endB: " << endB << endl
                        //      << endl;
                        thrust::transform(vA.begin() + beginA, vA.begin() + endA, vB.begin() + beginB, ponto.begin() + l, calculo(vA.data(), vB.data()));
                        l++;
                        // for (int x = 0; x < ponto.size(); x++)
                        // {
                        //     cout << "ponto: " << ponto[x] << endl;
                        // }
                    }
                    endB++;
                }
            }
        }
    }
    max = 0;

    cout
        << "Score Maximo: " << max << endl;
    return 0;
}