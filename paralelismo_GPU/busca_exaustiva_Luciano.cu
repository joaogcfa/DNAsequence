#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <iomanip>
#include <cstring>

using namespace std;

struct calculo
{
    thrust::device_ptr<char> vA;
    thrust::device_ptr<char> vB;

    calculo(thrust::device_ptr<char> vA_, thrust::device_ptr<char> vB_)
        : vA(vA_),
          vB(vB_)
    {
    }
    __host__ __device__

        int
        operator()(const int &i, const int &j)
    {
        int w;
        if (vA[i] == vB[j])
        {

            w = 2;
        }
        else
        {
            w = -1;
        }

        return w;
    }
};

// struct inline_score
// {

//     __host__ __device__ int
//     operator()(const int &i, const int &j)
//     {
//         int actual = j;
//         int left = i - 1;
//         int max;

//         if (left >= actual)
//             max = left;
//         else
//             max = actual;

//         if (max < 0)
//             max = 0;

//         return max;
//     }
// };

int main()
{

    int n, m, max, endA, endB, sizeSubA, sizeSubB;
    string seq1, seq2;

    cin >> n >> m;
    cin >> seq1 >> seq2;

    thrust::device_vector<char> vA(seq1.begin(), seq1.end());
    thrust::device_vector<char> vB(seq2.begin(), seq2.end());

    thrust::device_vector<int> ponto(10);

    cout << "seq1 size: " << n << endl;
    cout << "seq2 size: " << m << endl;
    cout << "seq1 is: " << seq1 << endl;
    cout << "seq2 is: " << seq2 << endl;

    for (int beginA = 0; beginA < vA.size(); beginA++)
    {
        for (int beginB = 0; beginB < vB.size(); beginB++)
        {
            endA = beginA;
            while (endA < vA.size())
            {
                endB = beginB;
                sizeSubA = endA - beginA;
                while (endB < vB.size())
                {
                    sizeSubB = endB - beginB;
                    if (sizeSubA == sizeSubB && beginB + sizeSubA < vB.size())
                    {
                        cout << "beginA: " << beginA << endl;
                        cout << "endA: " << endA << endl
                             << endl;
                        cout << "beginB: " << beginB << endl;
                        cout << "endB: " << endB << endl
                             << endl;
                        thrust::transform(vA.begin() + beginA, vA.begin() + endA, vB.begin() + beginB, ponto.begin(), calculo(vA.data(), vB.data()));
                        for (int x = 0; x < ponto.size(); x++)
                        {
                            cout << "ponto: " << ponto[x] << endl;
                        }
                        endA++;
                    }
                    endB++;
                }
            }
        }
    }
    max = 0;

    cout
        << "Score Maximo: " << max << endl;
    return 0;
}