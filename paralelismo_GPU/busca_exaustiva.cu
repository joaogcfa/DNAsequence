#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <iomanip>
#include <cstring>

using namespace std;

struct calculo
{
    thrust::device_ptr<char> vA;
    thrust::device_ptr<char> vB;

    calculo(thrust::device_ptr<char> vA_, thrust::device_ptr<char> vB_)
        : vA(vA_),
          vB(vB_)
    {
    }
    __host__ __device__

        int
        operator()(const int &i)
    {
        int w;
        if (vA[i] == vB[i])
        {

            w = 2;
        }
        else
        {
            w = -1;
        }

        return w;
    }
};

// struct inline_score
// {

//     __host__ __device__ int
//     operator()(const int &i, const int &j)
//     {
//         int actual = j;
//         int left = i - 1;
//         int max;

//         if (left >= actual)
//             max = left;
//         else
//             max = actual;

//         if (max < 0)
//             max = 0;

//         return max;
//     }
// };

int main()
{

    int n, m, max, endA, endB, sizeSubA, sizeSubB;
    string seq1, seq2;

    cin >> n >> m;
    cin >> seq1 >> seq2;

    thrust::device_vector<char> vA(seq1.begin(), seq1.end());
    thrust::device_vector<char> vB(seq2.begin(), seq2.end());

    thrust::device_vector<int> ponto;

    cout << "seq1 size: " << n << endl;
    cout << "seq2 size: " << m << endl;
    cout << "seq1 is: " << seq1 << endl;
    cout << "seq2 is: " << seq2 << endl;

    for (int beginA = 0; beginA < vA.size(); beginA++)
    {
        for (int beginB = 0; beginB < vB.size(); beginB++)
        {
            endA = beginA;
            while (endA < vA.size())
            {
                endB = beginB;
                sizeSubA = endA - beginA;
                while (endB < vB.size())
                {
                    sizeSubB = endB - beginB;
                    if (sizeSubA == sizeSubB && beginB + sizeSubA < vB.size())
                    {
                        cout << "i: " << beginA << endl;
                        cout << "h: " << endA << endl
                             << endl;
                        cout << "beginB: " << beginB << endl;
                        cout << "L: " << endB << endl
                             << endl;
                        thrust::transform(vA.begin() + beginA, vA.begin() + endA, vB.begin() + beginB, ponto.begin(), calculo(vA.data(), vB.data()));
                    }
                    endB++;
                }
                endA++;
            }
        }
    }
    max = 0;

    cout
        << "Score Maximo: " << max << endl;
    return 0;
}