#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <iomanip>
#include <cstring>

using namespace std;
typedef vector<vector<int>> matrix;

struct calculo
{
  thrust::device_ptr<char> d_S;
  char letradeT;
  thrust::device_ptr<int> calc;

  calculo(thrust::device_ptr<char> d_S_, char letradeT_, thrust::device_ptr<int> calc_)
      : d_S(d_S_),
        letradeT(letradeT_),
        calc(calc_)
  {
  }
  __host__ __device__

      int
      operator()(const int &i)
  {
    int w;
    int diagonal;
    int insertion;
    int max = 0;
    if (d_S[i - 1] == letradeT)
    {
      w = 2;
    }
    else
    {
      w = -1;
    }

    diagonal = calc[i - 1] + w;
    insertion = calc[i] - 1;

    if (diagonal > max)
    {
      max = diagonal;
    }
    if (insertion > max)
    {
      max = insertion;
    }

    return max;
  }
};
struct inline_score
{

  __host__ __device__ int
  operator()(const int &i, const int &j)
  {
    int actual = j;
    int left = i - 1;
    int max;

    if (left >= actual)
      max = left;
    else
      max = actual;

    if (max < 0)
      max = 0;

    return max;
  }
};
vector<string> subGenerator(int size, string seq)
{
  vector<string> sub_seqs;
  for (int i = 0; i < size; i++)
  {
    string cand = "";
    for (int j = i; j < size; j++)
    {
      cand += seq[j];
      sub_seqs.push_back(cand);
    }
  }

  return sub_seqs;
};

int main()
{

  int N, M,n,m, score,maxlc, max = 0;
  string S, T, seq1, seq2;
  vector<string> subs1, subs2;
  matrix H;

  cin >> n >> m;
  cin >> seq1 >> seq2;

  // cout << "seq1 size: " << n << endl;
  // cout << "seq2 size: " << m << endl;
  // cout << "seq1 is: " << seq1 << endl;
  // cout << "seq2 is: " << seq2 << endl;

  subs1 = subGenerator(n, seq1);

  // thrust::device_vector<string> sub_device(subs1);
  // thrust::copy(subs1.begin(), subs1.end(), sub_vector1.begin());

  subs2 = subGenerator(m, seq2);
  // thrust::copy(thrust::device, subs2.begin(), subs2.end(), sub_vector2.begin());

  for (int i = 0; i < int(subs1.size()); i++)
  {
    for (int j = 0; j < int(subs2.size()); j++)
    {

      string S = subs1[i];
      string T = subs2[j];


      N = S.size();
      M = T.size();

      thrust::device_vector<int> calc(N + 1);
      thrust::device_vector<int> temp(N + 1);

      thrust::fill(calc.begin(), calc.end(), 0);
      thrust::fill(temp.begin(), temp.end(), 0);

      thrust::device_vector<char> d_S(S.begin(), S.end());

      thrust::counting_iterator<int> c0(1);
      thrust::counting_iterator<int> c1(N + 1);

      for (int j = 0; j < M; j++)
      {
        char letradeT = T[j];
        thrust::transform(c0, c1, temp.begin() + 1, calculo(d_S.data(), letradeT, calc.data()));
        thrust::inclusive_scan(temp.begin() + 1, temp.end(), calc.begin() + 1, inline_score());
        // for (int l = 0; l < calc.size(); l++)
        // {
        //   cout << " " << calc[l];
        // }
        score = thrust::reduce(calc.begin(), calc.end(), 0, thrust::maximum<int>());

        if (score > maxlc)
        {
          maxlc = score;
          if (maxlc > max)
          max = maxlc;
        }
        cout << endl;
      }
    }
    cout << "*" << endl;
  }
  cout << "Score Maximo: " << max << endl;
  return 0;
}
// cout << endl;
// cout << "Sub Seq1 Best: " << seq1Def.substr(1, sizeof(seq1Def)) << endl;
// cout << "Sub Seq2 Best: " << seq2Def.substr(1, sizeof(seq2Def)) << endl;
// }