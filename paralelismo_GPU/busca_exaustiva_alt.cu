#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <iomanip>
#include <cstring>

using namespace std;

struct calculo
{
    thrust::device_ptr<char> vA;
    thrust::device_ptr<char> vB;

    calculo(thrust::device_ptr<char> vA_, thrust::device_ptr<char> vB_)
        : vA(vA_),
          vB(vB_)
    {
    }
    __host__ __device__

        int
        operator()(const int &i, const int &j)
    {
        int w = -1;
        if (vA[i] == vB[j])
            w = 2;
        return w;
    }
};

// struct inline_score
// {

//     __host__ __device__ int
//     operator()(const int &i, const int &j)
//     {
//         int actual = j;
//         int left = i - 1;
//         int max;

//         if (left >= actual)
//             max = left;
//         else
//             max = actual;

//         if (max < 0)
//             max = 0;

//         return max;
//     }
// };

int main()
{

    int n, m, score, endA, endB, sizeSubA, sizeSubB, tam;
    string seq1, seq2;

    cin >> n >> m;
    cin >> seq1 >> seq2;

    thrust::device_vector<char> vA(seq1.begin(), seq1.end());
    thrust::device_vector<char> vB(seq2.begin(), seq2.end());

    if (seq1.size() > seq2.size())
    {
        tam = seq1.size();
    }
    else
    {
        tam = seq2.size();
    }

    cout << tam << endl;

    thrust::device_vector<int> ponto(tam);

    for (int x = 0; x < ponto.size(); x++)
    {
        cout << "ponto: " << ponto[x] << endl;
    }

    thrust::fill(ponto.begin(), ponto.end(), 0);

    cout << "seq1 size: " << n << endl;
    cout << "seq2 size: " << m << endl;
    cout << "seq1 is: " << seq1 << endl;
    cout << "seq2 is: " << seq2 << endl;

    for (int beginA = 0; beginA < vA.size(); beginA++)
    {
        for (int beginB = 0; beginB < vB.size(); beginB++)
        {
            endA = beginA;
            while (endA < vA.size())
            {
                endB = beginB;
                endA++;
                sizeSubA = endA - beginA;
                while (endB < vB.size())
                {
                    endB++;
                    sizeSubB = endB - beginB;
                    if (sizeSubA == sizeSubB && beginB + sizeSubA < vB.size())
                    {
                        // cout << "beginA: " << beginA << endl;
                        // cout << "endA: " << endA << endl
                        //      << endl;
                        // cout << "beginB: " << beginB << endl;
                        // cout << "endB: " << endB << endl
                        //      << endl;
                        thrust::transform(vA.begin() + beginA, vA.begin() + endA, vB.begin() + beginB, ponto.begin(), calculo(vA.data(), vB.data()));
                        for (int x = 0; x < ponto.size(); x++)
                        {
                            cout << "ponto: " << ponto[x] << endl;
                        }
                    }
                }
            }
        }
    }

    // cout << "Score Maximo: " << score << endl;
    return 0;
}